#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#include "../include/constants.h"
#include "../include/histogram_kernels.h"

using namespace std;

__global__ void init_DpE_kernel(RealTypeP *DpE, int N_PART, int BINS) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= 2 * BINS) return;

    double numerator = 6.0E-26 * N_PART;
    double denominator = SIGMA_VELOCITY * sqrt(2.0 * M_PI);
    double exponent = -pow(3.0e-23 * (1.0 * i / BINS - 0.999) / SIGMA_VELOCITY, 2) / 2;
    DpE[i] = (numerator / denominator) * exp(exponent);
}

__global__ void init_DxE_kernel(RealTypeX *d_DxE, int N_PART, int BINS) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= (BINS + 1) << 1) return;
    if (i < 2) {
        // Maybe just i == 0?
        d_DxE[0] = 0.0;
        d_DxE[1] = 0.0;
        d_DxE[2 * BINS + 2] = 0.0;
        d_DxE[2 * BINS + 3] = 0.0;
    }
    d_DxE[i] = 1.0E-3 * N_PART;
}

__device__ uint32_t generate_random(uint32_t base_seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t seed = base_seed + idx + blockIdx.x + threadIdx.x;

    hiprandState state;
    hiprand_init(seed, idx, 5, &state);

    int random = hiprand(&state); 
    return random;
}

__device__ uint32_t xorshift32(uint32_t* seed) {
    uint32_t x = *seed;
    x ^= x << 13;
    x ^= x >> 17;
    x ^= x << 5;
    *seed = x;
    return x;
}

__device__ double d_xorshift(uint32_t *seed) {
    uint32_t x = xorshift32(seed);
    return (double)x / (double)UINT32_MAX;
}

// __device__ double f_xorshift(uint32_t *seed) {
//     uint32_t x = xorshift32(seed);
//     return (double)x / (double)UINT32_MAX;
// }

__global__ void init_x_kernel(RealTypeX* d_x, uint32_t base_seed, int N_PART) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N_PART) return;

    uint32_t seed = generate_random(base_seed);
    d_x[idx] = d_xorshift(&seed) * 0.5;
}

__global__ void init_p_kernel(RealTypeP* d_p,  uint32_t base_seed, int N_PART) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= (N_PART >> 1)) return;
    
    uint32_t seed = generate_random(base_seed);

    // Generate two random values using XORShift
    double randomValue1 = d_xorshift(&seed);
    double randomValue2 = d_xorshift(&seed);

    // Box-Muller transform to generate two normally distributed random numbers
    double xi1 = sqrt(-2.0 * log(randomValue1 + EPSILON));
    double xi2 = 2.0 * M_PI * randomValue2;

    // Store the generated values in the p array
    d_p[2 * idx] = xi1 * cos(xi2) * SIGMA_VELOCITY;
    d_p[2 * idx + 1] = xi1 * sin(xi2) * SIGMA_VELOCITY;
    
}

__global__ void update_histograms_kernel(RealTypeX *d_x, RealTypeP *d_p, int *h, int *g, int *hg, int N_PART, int BINS) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N_PART) return;

    // Calculate histogram indices based on particle data
    int h_idx = static_cast<int>(floor((d_x[idx] + 0.5) * (1.99999999999999 * BINS) + 2.0));
    int g_idx = static_cast<int>(floor((d_p[idx] / 3.0e-23 + 1) * (0.999999999999994 * BINS)));

    int hg_idx = (2 * BINS) * h_idx + g_idx;

    // Use atomic operations to avoid race conditions when updating shared memory
    atomicAdd(&h[h_idx], 1);
    atomicAdd(&g[g_idx], 1);
    atomicAdd(&hg[hg_idx], 1);
}

// Kernel function to update positions and momenta
__global__ void simulate_particle_motion(int number_of_steps, RealTypeX *d_x, RealTypeP *d_p, int N_PART, RealTypeConstant DT, RealTypeConstant M, RealTypeConstant sigmaL) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N_PART) return;

    uint32_t seed = idx + blockIdx.x + threadIdx.x * 31;

    // Change to double for x_tmp since x is now double
    RealTypeX x_tmp = d_x[idx];
    RealTypeP p_tmp = d_p[idx];  // Keep p_tmp as double

    int signop, k;
    double deltaX;

    // Main particle loop
    for (int step = 0; step < number_of_steps; ++step) {
        x_tmp += p_tmp * (DT / M);
        signop = copysign(1.0, p_tmp);
        k = trunc(x_tmp + 0.5 * signop);

        if (k == 0) continue;

        deltaX = sqrt(0.4) * sqrt(-2.0 * log(d_xorshift(&seed) + EPSILON)) * cos(2.0 * M_PI * d_xorshift(&seed)) * sigmaL;
        deltaX = (fabs(deltaX) > 1.0 ? 1.0 * copysign(1.0, deltaX) : deltaX);
        x_tmp = (k % 2 ? -1.0 : 1.0) * (x_tmp - k) + deltaX;

        if (fabs(x_tmp) > 0.502) {
            x_tmp = 1.004 * copysign(1.0, x_tmp) - x_tmp;
        }
        p_tmp = fabs(p_tmp);

        for (int l = 1; l <= labs(k); ++l) {
            // p_tmp = sqrt(max(0.0, p_tmp * p_tmp + ALFA * (p_tmp - PMIN) * (PMAX - p_tmp) * (d_xorshift(&seed) - 0.5)));
            p_tmp = sqrt(p_tmp * p_tmp + ALFA * (p_tmp - PMIN) * (PMAX - p_tmp) * (d_xorshift(&seed) - 0.5));
        }
        p_tmp *= (k % 2 ? -1.0 : 1.0) * signop;
    }
    // Update global memory
    d_x[idx] = x_tmp;
    d_p[idx] = p_tmp;
}

// CUDA kernel for energy sum calculation
__global__ void energy_sum_kernel(RealTypeP *d_p, RealTypePartialSum *partialSum, int N_PART) {
    extern __shared__ double sharedData[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load data into shared memory
    sharedData[tid] = (i < N_PART) ? d_p[i] * d_p[i] : 0.0;
    __syncthreads();
    
    // Perform reduction within each block
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sharedData[tid] += sharedData[tid + s];
        }
        __syncthreads();
    }
    
    // Write result of this block's partial sum to global memory
    if (tid == 0) {
        partialSum[blockIdx.x] = sharedData[0];
    }
}
