#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#include "../include/constants.h"
#include "../include/histogram_kernels.h"

using namespace std;

__global__ void init_DpE_kernel(double *DpE, int N_PART, int BINS) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= 2 * BINS) return;

    double numerator = 6.0E-26 * N_PART;
    double denominator = SIGMA_VELOCITY * sqrt(2.0 * M_PI);
    double exponent = -pow(3.0e-23 * (1.0 * i / BINS - 0.999) / SIGMA_VELOCITY, 2) / 2;
    DpE[i] = (numerator / denominator) * exp(exponent);
}

__global__ void init_DxE_kernel(float *d_DxE, int N_PART, int BINS) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= (BINS + 1) << 1) return;
    if (i < 2) {
        // Maybe just i == 0?
        d_DxE[0] = 0.0f;
        d_DxE[1] = 0.0f;
        d_DxE[2 * BINS + 2] = 0.0f;
        d_DxE[2 * BINS + 3] = 0.0f;
    }
    d_DxE[i] = 1.0E-3f * N_PART;
}

__device__ uint32_t generate_random(uint32_t base_seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t seed = base_seed + idx + blockIdx.x + threadIdx.x;

    hiprandState state;
    hiprand_init(seed, idx, 5, &state);

    int random = hiprand(&state); 
    return random;
}

__device__ uint32_t xorshift32(uint32_t* seed) {
    uint32_t x = *seed;
    x ^= x << 13;
    x ^= x >> 17;
    x ^= x << 5;
    *seed = x;
    return x;
}

__device__ double d_xorshift(uint32_t *seed) {
    uint32_t x = xorshift32(seed);
    return (double)x / (double)UINT32_MAX;
}

__device__ float f_xorshift(uint32_t *seed) {
    uint32_t x = xorshift32(seed);
    return (float)x / (float)UINT32_MAX;
}

__global__ void init_x_kernel(float* d_x, uint32_t base_seed, int N_PART) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N_PART) return;

    uint32_t seed = generate_random(base_seed);
    d_x[idx] = f_xorshift(&seed) * 0.5f;
}

__global__ void init_p_kernel(double* d_p,  uint32_t base_seed, int N_PART) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= (N_PART >> 1)) return;
    
    uint32_t seed = generate_random(base_seed);

    // Generate two random values using XORShift
    double randomValue1 = d_xorshift(&seed);
    double randomValue2 = d_xorshift(&seed);

    // Box-Muller transform to generate two normally distributed random numbers
    double xi1 = sqrt(-2.0 * log(randomValue1 + EPSILON));
    double xi2 = 2.0 * M_PI * randomValue2;

    // Store the generated values in the p array
    d_p[2 * idx] = xi1 * cos(xi2) * SIGMA_VELOCITY;
    d_p[2 * idx + 1] = xi1 * sin(xi2) * SIGMA_VELOCITY;
    
}

__global__ void update_histograms_kernel(float *d_x, double *d_p, int *h, int *g, int *hg, int N_PART, int BINS) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N_PART) return;

    // Calculate histogram indices based on particle data
    int h_idx = static_cast<int>(floorf((d_x[idx] + 0.5f) * (1.99999999999999f * BINS) + 2.0f));
    int g_idx = static_cast<int>(floor((d_p[idx] / 3.0e-23 + 1) * (0.999999999999994 * BINS)));

    int hg_idx = (2 * BINS) * h_idx + g_idx;

    // Use atomic operations to avoid race conditions when updating shared memory
    atomicAdd(&h[h_idx], 1);
    atomicAdd(&g[g_idx], 1);
    atomicAdd(&hg[hg_idx], 1);
}

// Kernel function to update positions and momenta
__global__ void simulate_particle_motion(int number_of_steps, float *d_x, double *d_p, int N_PART, float DT, float M, float sigmaL) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N_PART) return;

    uint32_t seed = idx + blockIdx.x + threadIdx.x * 31;

    // Change to float for x_tmp since x is now float
    float x_tmp = d_x[idx];
    double p_tmp = d_p[idx];  // Keep p_tmp as double

    int signop, k;
    float deltaX;

    // Main particle loop
    for (int step = 0; step < number_of_steps; ++step) {
        x_tmp += p_tmp * (DT / M);
        signop = copysign(1.0, p_tmp);
        k = truncf(x_tmp + 0.5f * signop);

        if (k == 0) continue;

        // float xi1 = sqrtf(-2.0f * logf(f_xorshift(&seed) + EPSILON));
        // float xi2 = 2.0f * (float)M_PI * f_xorshift(&seed);
        deltaX = sqrtf(fabsf(k)) * sqrtf(-2.0f * logf(f_xorshift(&seed) + EPSILON)) * cosf(2.0f * (float)M_PI * f_xorshift(&seed)) * sigmaL;  // Use float functions
        deltaX = (fabsf(deltaX) > 1.0f ? 1.0f * copysignf(1.0f, deltaX) : deltaX);
        x_tmp = (k % 2 ? -1.0f : 1.0f) * (x_tmp - k) + deltaX;

        if (fabsf(x_tmp) > 0.502f) {
            x_tmp = 1.004f * copysignf(1.0f, x_tmp) - x_tmp;
        }
        p_tmp = fabs(p_tmp);

        for (int l = 1; l <= labs(k); ++l) {
            // float DeltaE = ALFA * (p_tmp - PMIN) * (PMAX - p_tmp);
            // double value = p_tmp * p_tmp + DeltaE * (f_xorshift(&seed) - 0.5);
            // if (value < 0) {
            //     value = 0;
            // }
            p_tmp = sqrt(max(0.0f, p_tmp * p_tmp + ALFA * (p_tmp - PMIN) * (PMAX - p_tmp) * (f_xorshift(&seed) - 0.5)));
        }
        p_tmp *= (k % 2 ? -1.0 : 1.0) * signop;
    }
    // Update global memory
    d_x[idx] = x_tmp;
    d_p[idx] = p_tmp;
}

// CUDA kernel for energy sum calculation
__global__ void energy_sum_kernel(double *d_p, float *partialSum, int N_PART) {
    extern __shared__ double sharedData[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load data into shared memory
    sharedData[tid] = (i < N_PART) ? d_p[i] * d_p[i] : 0.0;
    __syncthreads();
    
    // Perform reduction within each block
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sharedData[tid] += sharedData[tid + s];
        }
        __syncthreads();
    }
    
    // Write result of this block's partial sum to global memory
    if (tid == 0) {
        partialSum[blockIdx.x] = sharedData[0];
    }
}
