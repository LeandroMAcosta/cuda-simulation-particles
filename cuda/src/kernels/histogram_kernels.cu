#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#include "../include/constants.h"
#include "../include/histogram_kernels.h"

using namespace std;

__global__ void init_DpE_kernel(double *DpE, int N_PART, int BINS) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= 2 * BINS) return;

    double numerator = 6.0E-26 * N_PART;
    double denominator = 5.24684E-24 * sqrt(2.0 * M_PI);
    double exponent = -pow(3.0e-23 * (1.0 * i / BINS - 0.999) / 5.24684E-24, 2) / 2;
    DpE[i] = (numerator / denominator) * exp(exponent);
}

__global__ void init_DxE_kernel(double *DxE, int N_PART, int BINS) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= (BINS + 1) << 1) return;
    if (i < 2) {
        // Maybe just i == 0?
        DxE[0] = 0.0;
        DxE[1] = 0.0;
        DxE[2 * BINS + 2] = 0.0;
        DxE[2 * BINS + 3] = 0.0;
    }
    DxE[i] = 1.0E-3 * N_PART;
}

__device__ uint32_t generate_random(uint32_t base_seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t seed = base_seed + idx + blockIdx.x + threadIdx.x;

    hiprandState state;
    hiprand_init(seed, idx, 5, &state);

    int random = hiprand(&state); 
    return random;
}

__device__ uint32_t xorshift32(uint32_t* seed) {
    uint32_t x = *seed;
    x ^= x << 13;
    x ^= x >> 17;
    x ^= x << 5;
    *seed = x;
    return x;
}

__device__ double d_xorshift(uint32_t *seed) {
    uint32_t x = xorshift32(seed);
    return (double)x / (double)UINT32_MAX;
}

__device__ float f_xorshift(uint32_t *seed) {
    uint32_t x = xorshift32(seed);
    return (float)x / (float)UINT32_MAX;
}

__global__ void init_x_kernel(float* x, uint32_t base_seed, int N_PART) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N_PART) return;

    uint32_t seed = generate_random(base_seed);
    x[idx] = f_xorshift(&seed) * 0.5f;
}

__global__ void init_p_kernel(double* d_p,  uint32_t base_seed, int N_PART) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= (N_PART >> 1)) return;
    
    uint32_t seed = generate_random(base_seed);

    // Generate two random values using XORShift
    double randomValue1 = (double)(xorshift32(&seed)) / UINT32_MAX;
    double randomValue2 = (double)(xorshift32(&seed)) / UINT32_MAX;

    // Box-Muller transform to generate two normally distributed random numbers
    double xi1 = sqrt(-2.0 * log(randomValue1 + 1E-35));
    double xi2 = 2.0 * M_PI * randomValue2;

    // Store the generated values in the p array
    d_p[2 * idx] = xi1 * cos(xi2) * 5.24684E-24;
    d_p[2 * idx + 1] = xi1 * sin(xi2) * 5.24684E-24;
    
}

__global__ void update_histograms_kernel(float *x, double *d_p, int *h, int *g, int *hg, int N_PART, int BINS) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N_PART) return;

    // Calculate histogram indices based on particle data
    int h_idx = static_cast<int>(floorf((x[idx] + 0.5f) * (1.99999999999999f * BINS) + 2.0f));
    int g_idx = static_cast<int>(floor((d_p[idx] / 3.0e-23 + 1) * (0.999999999999994 * BINS)));

    int hg_idx = (2 * BINS) * h_idx + g_idx;

    // Use atomic operations to avoid race conditions when updating shared memory
    atomicAdd(&h[h_idx], 1);
    atomicAdd(&g[g_idx], 1);
    atomicAdd(&hg[hg_idx], 1);
}

// Kernel function to update positions and momenta
__global__ void simulate_particle_motion(int number_of_steps, float *x, double *d_p, int N_PART, float DT, double M, float sigmaL, float alfa, float pmin, float pmax) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N_PART) return;

    uint32_t seed = idx + blockIdx.x + threadIdx.x * 31;

    // Change to float for x_tmp since x is now float
    float x_tmp = x[idx];
    double p_tmp = d_p[idx];  // Keep p_tmp as double

    int signop, k;

    // Main particle loop
    for (int step = 0; step < number_of_steps; ++step) {
        // Operations with x_tmp now involve float
        x_tmp += p_tmp * (DT / M);  // Cast DT / M to float
        // x_tmp += (float) (p_tmp * (DT / M));  // Cast DT / M to float
        signop = copysign(1.0, p_tmp);  // Keep signop as double
        k = truncf(x_tmp + 0.5f * signop);  // Use truncf for float

        if (k == 0) continue;

        float randomValue = f_xorshift(&seed);
        float xi1 = sqrtf(-2.0f * logf(randomValue + 1E-35f));
        randomValue = f_xorshift(&seed);
        float xi2 = 2.0f * (float)M_PI * randomValue;
        float deltaX = sqrtf(fabsf(k)) * xi1 * cosf((float)xi2) * sigmaL;  // Use float functions

        deltaX = (fabsf(deltaX) > 1.0f ? 1.0f * copysignf(1.0f, deltaX) : deltaX);
        x_tmp = (k % 2 ? -1.0f : 1.0f) * (x_tmp - k) + deltaX;

        if (fabsf(x_tmp) > 0.502f) {
            x_tmp = 1.004f * copysignf(1.0f, x_tmp) - x_tmp;
        }
        p_tmp = fabs(p_tmp);  // Keep p_tmp operations in double

        for (int l = 1; l <= labs(k); ++l) {
            float DeltaE = alfa * (p_tmp - pmin) * (pmax - p_tmp);
            randomValue = f_xorshift(&seed);
            double value = p_tmp * p_tmp + DeltaE * (randomValue - 0.5);
            if (value < 0) {
                value = 0;
            }
            p_tmp = sqrt(value);
        }
        p_tmp *= (k % 2 ? -1.0 : 1.0) * signop;
    }
    // Update global memory
    x[idx] = x_tmp;  // Store the float result back to x
    d_p[idx] = p_tmp;  // Store the double result back to p
}

// CUDA kernel for energy sum calculation
__global__ void energy_sum_kernel(double *d_p, double *partialSum, int N_PART) {
    extern __shared__ double sharedData[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load data into shared memory
    sharedData[tid] = (i < N_PART) ? d_p[i] * d_p[i] : 0.0;
    __syncthreads();
    
    // Perform reduction within each block
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sharedData[tid] += sharedData[tid + s];
        }
        __syncthreads();
    }
    
    // Write result of this block's partial sum to global memory
    if (tid == 0) {
        partialSum[blockIdx.x] = sharedData[0];
    }
}
