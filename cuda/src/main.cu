#include <cmath>
#include <cstring>
#include <cstdio>
// #include <omp.h_h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "./include/utils.h"
#include "./include/histogram_kernels.h"

using namespace std;

int main()
{
    int N_THREADS = 0, N_PART = 0, BINS = 0;
    bool resume, dump;

    unsigned int Ntandas = 0u;
    char inputFilename[255], saveFilename[255];
    
    float M;
    float DT; 
    float sigmaL = 0.0f;

    int X0 = 1;
    char filename[32];

    unsigned int evolution = 0u;
    // float alfa = 1.0e-4f;
    // float pmin = 2.0E-026f, pmax = 3.0E-023f;

    int steps[500];
    char data_filename[] = "datos.in";
    load_parameters_from_file(data_filename, &N_PART, &BINS, &DT, &M, &N_THREADS, &Ntandas, steps, inputFilename,
                              saveFilename, &resume, &dump, &sigmaL);

    printf("Parameters loaded from file:\n");
    printf("N_PART=%d\n", N_PART);
    printf("BINS=%d\n", BINS);
    printf("DT=%.60f\n", DT);
    printf("M=%.60f\n", M);
    printf("N_THREADS=%d\n", N_THREADS);
    printf("Ntandas=%d\n", Ntandas);
    printf("inputFilename=%s\n", inputFilename);
    printf("saveFilename=%s\n", saveFilename);
    printf("resume=%d\n", resume);
    printf("dump=%d\n", dump);
    printf("sigmaL=%f\n", sigmaL);

    // Unified Memory Allocation for arrays using hipMallocManaged
    double *h_p;
    float *h_x;
    h_x = (float *)malloc(sizeof(float) * N_PART);
    h_p = (double *)malloc(sizeof(double) * N_PART);

    double *d_p, *d_DpE;
    float *d_x, *d_DxE;
    hipMalloc(&d_x, sizeof(float) * N_PART);
    hipMalloc(&d_p, sizeof(double) * N_PART);
    hipMalloc(&d_DxE, sizeof(float) * (2 * BINS + 4));
    hipMalloc(&d_DpE, sizeof(double) * (2 * BINS));

    // Launch CUDA kernel for parallel d_DpE computation
    int threadsPerBlock = 512;

    int blocksPerGridForDpE = (2 * BINS + threadsPerBlock - 1) / threadsPerBlock;
    init_DpE_kernel<<<blocksPerGridForDpE, threadsPerBlock>>>(d_DpE, N_PART, BINS);

    int blocksPerGridForDxE = (2 * BINS + threadsPerBlock - 1) / threadsPerBlock;
    init_DxE_kernel<<<blocksPerGridForDxE, threadsPerBlock>>>(d_DxE, N_PART, BINS);
  
    // Host arrays (used in CPU)
    int *h_h, *h_g, *h_hg;

    // Initialize host arrays with zeros
    h_h = (int *)calloc(2 * BINS + 4, sizeof(int));
    h_g = (int *)calloc(2 * BINS, sizeof(int));
    h_hg = (int *)calloc((2 * BINS + 4) * (2 * BINS), sizeof(int));

    // Device arrays (used in GPU)
    int *d_h, *d_g, *d_hg;
    hipMalloc(&d_h, sizeof(int) * (2 * BINS + 4));
    hipMalloc(&d_g, sizeof(int) * (2 * BINS));
    hipMalloc(&d_hg, sizeof(int) * (2 * BINS + 4) * (2 * BINS));

    hipMemset(d_h, 0, (2 * BINS + 4) * sizeof(int));
    hipMemset(d_g, 0, (2 * BINS) * sizeof(int));
    hipMemset(d_hg, 0, (2 * BINS + 4) * (2 * BINS) * sizeof(int));

    // Check for resume condition
    if (!resume) {
        while (X0 == 1) {
            uint32_t base_seed_1 = static_cast<uint32_t>(time(NULL));
            uint32_t base_seed_2 = static_cast<uint32_t>(time(NULL) + 1);

            int numBlocksInitX = (N_PART + threadsPerBlock - 1) / threadsPerBlock;
            init_x_kernel<<<numBlocksInitX, threadsPerBlock>>>(d_x, base_seed_1, N_PART);

            int numBlocksInitP = ((N_PART >> 1) + threadsPerBlock - 1) / threadsPerBlock;
            init_p_kernel<<<numBlocksInitP, threadsPerBlock>>>(d_p, base_seed_2, N_PART);

            // The kernel  update_histograms_kernel uses d_x and p arrays to update h_h, h_g, h_hg arrays, so we need to synchronize.
            hipDeviceSynchronize();

            int numBlocksUpdateHist = (N_PART + threadsPerBlock - 1) / threadsPerBlock;
            update_histograms_kernel<<<numBlocksUpdateHist, threadsPerBlock>>>(d_x, d_p, d_h, d_g, d_hg, N_PART, BINS);
            hipDeviceSynchronize();

            float Et = energy_sum(d_p, N_PART, evolution, M);
            X0 = make_hist(h_h, h_g, h_hg, d_h, d_g, d_hg, d_DxE, d_DpE, "X0000000.dat", BINS, Et);
            if (X0 == 1) {
                cout << "Falló algún chi2: X0=" << X0 << endl;
            }
        }
    } else {
        read_data(inputFilename, h_x, h_p, &evolution, N_PART);
        hipMemcpy(d_x, h_x, sizeof(float) * N_PART, hipMemcpyHostToDevice);
        hipMemcpy(d_p, h_p, sizeof(double) * N_PART, hipMemcpyHostToDevice);
    }

    float Et = energy_sum(d_p, N_PART, evolution, M);
    cout << "PMIN=" << scientific << PMIN << " ALFA=" << ALFA << " Et=" << Et << endl;

    // Main loop to iterate through Ntandas
    for (unsigned int j = 0; j < Ntandas; j++) {
        // Kernel launch parameters
        int numBlocks = (N_PART + threadsPerBlock - 1) / threadsPerBlock;

        simulate_particle_motion<<<numBlocks, threadsPerBlock>>>(steps[j], d_x, d_p, N_PART, DT, M, sigmaL);
        hipDeviceSynchronize();

        int numBlocksUpdateHist = (N_PART + threadsPerBlock - 1) / threadsPerBlock;
        update_histograms_kernel<<<numBlocksUpdateHist, threadsPerBlock>>>(d_x, d_p, d_h, d_g, d_hg, N_PART, BINS);

        evolution += steps[j];
        if (evolution < 10000000) {
            sprintf(filename, "X%07d.dat", evolution);
        } else {
            sprintf(filename, "X%1.3e.dat", static_cast<double>(evolution));
            char *e = static_cast<char*>(memchr(filename, 'e', 32)); // Explicit cast to char*
            if (e) {
                strcpy(e + 1, e + 3); // Adjusting the position after 'e'
            }
        }

        if (dump) {
            hipMemcpy(h_x, d_x, sizeof(float) * N_PART, hipMemcpyDeviceToHost);
            hipMemcpy(h_p, d_p, sizeof(double) * N_PART, hipMemcpyDeviceToHost);
            save_data(saveFilename, h_x, d_p, evolution, N_PART);
        }

        Et = energy_sum(d_p, N_PART, evolution, M);
        make_hist(h_h, h_g, h_hg, d_h, d_g, d_hg, d_DxE, d_DpE, filename, BINS, Et);
    }

    cout << "Completo evolution = " << evolution << endl;

    // Free memory
    hipFree(d_x);
    hipFree(d_p);
    hipFree(d_DxE);
    hipFree(d_DpE);
    hipFree(d_h);
    hipFree(d_g);
    hipFree(d_hg);
    
    // Check for any device errors (after synchronization)
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Failed: %s\n", hipGetErrorString(err));
        exit(1);
    }

    return 0;
}
