#include <cmath>
#include <cstring>
#include <cstdio>
// #include <omp.h_h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "./include/utils.h"
#include "./include/histogram_kernels.h"
#include "./include/types.h"

using namespace std;

int main() {
    int N_THREADS = 0, N_PART = 0, BINS = 0;
    bool resume, dump;

    unsigned int Ntandas = 0u;
    char inputFilename[255], saveFilename[255];
    
    RealTypeConstant M, DT, sigmaL;

    int X0 = 1;
    char filename[32];

    unsigned int evolution = 0u;

    int steps[500];
    char data_filename[] = "datos.in";
    load_parameters_from_file(data_filename, &N_PART, &BINS, &DT, &M, &N_THREADS, &Ntandas, steps, inputFilename,
                              saveFilename, &resume, &dump, &sigmaL);

    printf("Parameters loaded from file:\n");
    printf("N_PART=%d\n", N_PART);
    printf("BINS=%d\n", BINS);
    printf("DT=%.60f\n", DT);
    printf("M=%.60f\n", M);
    printf("N_THREADS=%d\n", N_THREADS);
    printf("Ntandas=%d\n", Ntandas);
    printf("inputFilename=%s\n", inputFilename);
    printf("saveFilename=%s\n", saveFilename);
    printf("resume=%d\n", resume);
    printf("dump=%d\n", dump);
    printf("sigmaL=%f\n", sigmaL);

    // Unified Memory Allocation for arrays using hipMallocManaged
    RealTypeX *h_x;
    RealTypeP *h_p;
    h_x = (RealTypeX *)malloc(sizeof(h_x[0]) * N_PART);
    h_p = (RealTypeP *)malloc(sizeof(h_p[0]) * N_PART);

    RealTypeP *d_p, *d_DpE;
    RealTypeX *d_x, *d_DxE;
    hipMalloc(&d_x, sizeof(d_x[0]) * N_PART);
    hipMalloc(&d_p, sizeof(d_p[0]) * N_PART);
    hipMalloc(&d_DxE, sizeof(d_DxE[0]) * (2 * BINS + 4));
    hipMalloc(&d_DpE, sizeof(d_DpE[0]) * (2 * BINS));

    // Launch CUDA kernel for parallel d_DpE computation
    int threadsPerBlock = 512;

    int blocksPerGridForDpE = (2 * BINS + threadsPerBlock - 1) / threadsPerBlock;
    init_DpE_kernel<<<blocksPerGridForDpE, threadsPerBlock>>>(d_DpE, N_PART, BINS);

    int blocksPerGridForDxE = (2 * BINS + threadsPerBlock - 1) / threadsPerBlock;
    init_DxE_kernel<<<blocksPerGridForDxE, threadsPerBlock>>>(d_DxE, N_PART, BINS);
  
    // Host arrays (used in CPU)
    int *h_h, *h_g, *h_hg;

    // Initialize host arrays with zeros
    h_h = (int *)calloc(2 * BINS + 4, sizeof(int));
    h_g = (int *)calloc(2 * BINS, sizeof(int));
    h_hg = (int *)calloc((2 * BINS + 4) * (2 * BINS), sizeof(int));

    // Device arrays (used in GPU)
    int *d_h, *d_g, *d_hg;
    hipMalloc(&d_h, sizeof(int) * (2 * BINS + 4));
    hipMalloc(&d_g, sizeof(int) * (2 * BINS));
    hipMalloc(&d_hg, sizeof(int) * (2 * BINS + 4) * (2 * BINS));

    hipMemset(d_h, 0, (2 * BINS + 4) * sizeof(int));
    hipMemset(d_g, 0, (2 * BINS) * sizeof(int));
    hipMemset(d_hg, 0, (2 * BINS + 4) * (2 * BINS) * sizeof(int));

    // Check for resume condition
    if (!resume) {
        while (X0 == 1) {
            uint32_t base_seed_1 = static_cast<uint32_t>(time(NULL));
            uint32_t base_seed_2 = static_cast<uint32_t>(time(NULL) + 1);

            int numBlocksInitX = (N_PART + threadsPerBlock - 1) / threadsPerBlock;
            init_x_kernel<<<numBlocksInitX, threadsPerBlock>>>(d_x, base_seed_1, N_PART);

            int numBlocksInitP = ((N_PART >> 1) + threadsPerBlock - 1) / threadsPerBlock;
            init_p_kernel<<<numBlocksInitP, threadsPerBlock>>>(d_p, base_seed_2, N_PART);

            // The kernel  update_histograms_kernel uses d_x and p arrays to update h_h, h_g, h_hg arrays, so we need to synchronize.
            hipDeviceSynchronize();

            int numBlocksUpdateHist = (N_PART + threadsPerBlock - 1) / threadsPerBlock;
            update_histograms_kernel<<<numBlocksUpdateHist, threadsPerBlock>>>(d_x, d_p, d_h, d_g, d_hg, N_PART, BINS);
            hipDeviceSynchronize();

            double Et = energy_sum(d_p, N_PART, evolution, M);
            X0 = make_hist(h_h, h_g, h_hg, d_h, d_g, d_hg, d_DxE, d_DpE, "X0000000.dat", BINS, Et);
            if (X0 == 1) {
                cout << "Falló algún chi2: X0=" << X0 << endl;
            }
        }
    } else {
        read_data(inputFilename, h_x, h_p, &evolution, N_PART);
        hipMemcpy(d_x, h_x, sizeof(d_x[0]) * N_PART, hipMemcpyHostToDevice);
        hipMemcpy(d_p, h_p, sizeof(d_p[0]) * N_PART, hipMemcpyHostToDevice);
    }

    double Et = energy_sum(d_p, N_PART, evolution, M);
    cout << "PMIN=" << scientific << PMIN << " ALFA=" << ALFA << " Et=" << Et << endl;

    // Main loop to iterate through Ntandas
    for (unsigned int j = 0; j < Ntandas; j++) {
        // Kernel launch parameters
        int numBlocks = (N_PART + threadsPerBlock - 1) / threadsPerBlock;

        simulate_particle_motion<<<numBlocks, threadsPerBlock>>>(steps[j], d_x, d_p, N_PART, DT, M, sigmaL);
        hipDeviceSynchronize();

        int numBlocksUpdateHist = (N_PART + threadsPerBlock - 1) / threadsPerBlock;
        update_histograms_kernel<<<numBlocksUpdateHist, threadsPerBlock>>>(d_x, d_p, d_h, d_g, d_hg, N_PART, BINS);

        evolution += steps[j];
        if (evolution < 10000000) {
            sprintf(filename, "X%07d.dat", evolution);
        } else {
            sprintf(filename, "X%1.3e.dat", static_cast<double>(evolution));
            char *e = static_cast<char*>(memchr(filename, 'e', 32)); // Explicit cast to char*
            if (e) {
                strcpy(e + 1, e + 3);
            }
        }

        if (dump) {
            hipMemcpy(h_x, d_x, sizeof(h_x[0]) * N_PART, hipMemcpyDeviceToHost);
            hipMemcpy(h_p, d_p, sizeof(h_p[0]) * N_PART, hipMemcpyDeviceToHost);
            save_data(saveFilename, h_x, d_p, evolution, N_PART);
        }

        Et = energy_sum(d_p, N_PART, evolution, M);
        make_hist(h_h, h_g, h_hg, d_h, d_g, d_hg, d_DxE, d_DpE, filename, BINS, Et);
    }

    cout << "Completo evolution = " << evolution << endl;

    // Free memory
    hipFree(d_x);
    hipFree(d_p);
    hipFree(d_DxE);
    hipFree(d_DpE);
    hipFree(d_h);
    hipFree(d_g);
    hipFree(d_hg);
    
    // Check for any device errors (after synchronization)
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Failed: %s\n", hipGetErrorString(err));
        exit(1);
    }

    return 0;
}
