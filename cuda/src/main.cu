#include <cmath>
#include <cstring>
#include <cstdio>
// #include <omp.h_h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "./include/utils.h"
#include "./include/histogram_kernels.h"

using namespace std;

int main()
{
    int N_THREADS = 0, N_PART = 0, BINS = 0;
    bool resume, dump;

    unsigned int Ntandas = 0u;
    char inputFilename[255], saveFilename[255];
    
    float M;
    float DT; 
    float sigmaL = 0.0f;

    int X0 = 1;
    char filename[32];

    unsigned int evolution = 0u;
    float alfa = 1.0e-4f;
    float pmin = 2.0E-026f, pmax = 3.0E-023f;

    int steps[500];
    // hipMallocManaged(&steps, sizeof(int) * 500);

    char data_filename[] = "datos.in";
    load_parameters_from_file(data_filename, &N_PART, &BINS, &DT, &M, &N_THREADS, &Ntandas, steps, inputFilename,
                              saveFilename, &resume, &dump, &sigmaL);

    printf("Parameters loaded from file:\n");
    printf("N_PART=%d\n", N_PART);
    printf("BINS=%d\n", BINS);
    printf("DT=%.60f\n", DT);
    printf("M=%.60f\n", M);
    printf("N_THREADS=%d\n", N_THREADS);
    printf("Ntandas=%d\n", Ntandas);
    printf("inputFilename=%s\n", inputFilename);
    printf("saveFilename=%s\n", saveFilename);
    printf("resume=%d\n", resume);
    printf("dump=%d\n", dump);
    printf("sigmaL=%f\n", sigmaL);

    // Unified Memory Allocation for arrays using hipMallocManaged
    double *h_p;
    float *h_x;
    h_x = (float *)malloc(sizeof(float) * N_PART);
    h_p = (double *)malloc(sizeof(double) * N_PART);

    double *d_p, *d_DxE, *d_DpE;
    float *d_x;
    hipMalloc(&d_x, sizeof(float) * N_PART);
    hipMalloc(&d_p, sizeof(double) * N_PART);
    hipMalloc(&d_DxE, sizeof(double) * (2 * BINS + 4));
    hipMalloc(&d_DpE, sizeof(double) * (2 * BINS));

    // Launch CUDA kernel for parallel d_DpE computation
    int threadsPerBlock = 512;

    int blocksPerGridForDpE = (2 * BINS + threadsPerBlock - 1) / threadsPerBlock;
    init_DpE_kernel<<<blocksPerGridForDpE, threadsPerBlock>>>(d_DpE, N_PART, BINS);

    int blocksPerGridForDxE = (2 * BINS + threadsPerBlock - 1) / threadsPerBlock;
    init_DxE_kernel<<<blocksPerGridForDxE, threadsPerBlock>>>(d_DxE, N_PART, BINS);
  
    // Host arrays (used in CPU)
    int *h_h, *h_g, *h_hg;

    // Initialize host arrays with zeros
    h_h = (int *)calloc(2 * BINS + 4, sizeof(int));
    h_g = (int *)calloc(2 * BINS, sizeof(int));
    h_hg = (int *)calloc((2 * BINS + 4) * (2 * BINS), sizeof(int));

    // Device arrays (used in GPU)
    int *d_h, *d_g, *d_hg;
    hipMalloc(&d_h, sizeof(int) * (2 * BINS + 4));
    hipMalloc(&d_g, sizeof(int) * (2 * BINS));
    hipMalloc(&d_hg, sizeof(int) * (2 * BINS + 4) * (2 * BINS));

    hipMemset(d_h, 0, (2 * BINS + 4) * sizeof(int));
    hipMemset(d_g, 0, (2 * BINS) * sizeof(int));
    hipMemset(d_hg, 0, (2 * BINS + 4) * (2 * BINS) * sizeof(int));

    // Check for resume condition
    if (!resume) {
        while (X0 == 1) {
            uint32_t base_seed_1 = static_cast<uint32_t>(time(NULL));
            uint32_t base_seed_2 = static_cast<uint32_t>(time(NULL) + 1);

            int numBlocksInitX = (N_PART + threadsPerBlock - 1) / threadsPerBlock;
            init_x_kernel<<<numBlocksInitX, threadsPerBlock>>>(d_x, base_seed_1, N_PART);

            int numBlocksInitP = ((N_PART >> 1) + threadsPerBlock - 1) / threadsPerBlock;
            init_p_kernel<<<numBlocksInitP, threadsPerBlock>>>(d_p, base_seed_2, N_PART);

            // The kernel  update_histograms_kernel uses d_x and p arrays to update h_h, h_g, h_hg arrays, so we need to synchronize.
            hipDeviceSynchronize();

            int numBlocksUpdateHist = (N_PART + threadsPerBlock - 1) / threadsPerBlock;
            update_histograms_kernel<<<numBlocksUpdateHist, threadsPerBlock>>>(d_x, d_p, d_h, d_g, d_hg, N_PART, BINS);
            hipDeviceSynchronize();

            float Et = energy_sum(d_p, N_PART, evolution, M);
            X0 = make_hist(h_h, h_g, h_hg, d_h, d_g, d_hg, d_DxE, d_DpE, "X0000000.dat", BINS, Et);
            if (X0 == 1) {
                cout << "Falló algún chi2: X0=" << X0 << endl;
            }
        }
    } else {
        read_data(inputFilename, h_x, h_p, &evolution, N_PART);
        hipMemcpy(d_x, h_x, sizeof(float) * N_PART, hipMemcpyHostToDevice);
        hipMemcpy(d_p, h_p, sizeof(double) * N_PART, hipMemcpyHostToDevice);
    }

    float Et = energy_sum(d_p, N_PART, evolution, M);
    cout << "pmin=" << scientific << pmin << " alfa=" << alfa << " Et=" << Et << endl;

    // Main loop to iterate through Ntandas
    for (unsigned int j = 0; j < Ntandas; j++) {
        // Kernel launch parameters
        int numBlocks = (N_PART + threadsPerBlock - 1) / threadsPerBlock;

        simulate_particle_motion<<<numBlocks, threadsPerBlock>>>(steps[j], d_x, d_p, N_PART, DT, M, sigmaL, alfa, pmin, pmax);
        hipDeviceSynchronize();

        int numBlocksUpdateHist = (N_PART + threadsPerBlock - 1) / threadsPerBlock;
        update_histograms_kernel<<<numBlocksUpdateHist, threadsPerBlock>>>(d_x, d_p, d_h, d_g, d_hg, N_PART, BINS);

        evolution += steps[j];
        if (evolution < 10000000) {
            sprintf(filename, "X%07d.dat", evolution);
        } else {
            sprintf(filename, "X%1.3e.dat", static_cast<double>(evolution));
            char *e = static_cast<char*>(memchr(filename, 'e', 32)); // Explicit cast to char*
            if (e) {
                strcpy(e + 1, e + 3); // Adjusting the position after 'e'
            }
        }

        if (dump) {
            hipMemcpy(h_x, d_x, sizeof(float) * N_PART, hipMemcpyDeviceToHost);
            hipMemcpy(h_p, d_p, sizeof(double) * N_PART, hipMemcpyDeviceToHost);
            save_data(saveFilename, h_x, d_p, evolution, N_PART);
        }

        Et = energy_sum(d_p, N_PART, evolution, M);
        make_hist(h_h, h_g, h_hg, d_h, d_g, d_hg, d_DxE, d_DpE, filename, BINS, Et);
    }

    cout << "Completo evolution = " << evolution << endl;

    // Free memory
    hipFree(d_x);
    hipFree(d_p);
    hipFree(d_DxE);
    hipFree(d_DpE);
    hipFree(d_h);
    hipFree(d_g);
    hipFree(d_hg);
    
    // Check for any device errors (after synchronization)
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Failed: %s\n", hipGetErrorString(err));
        exit(1);
    }

    return 0;
}

/* para graficar en el gnuplot: (sacando un archivo "hists.eps")
set terminal postscript enhanced color eps 20
set output "hists.eps"
# histograma de d_x  (las dos líneas siguientes alcanzan para graficar las d_x
dentro del gnuplot) set style fill solid 1.0 # o medio transparente: set style
fill transparent solid 0.5 noborder set key left ; set xrange[-0.5:0.5] p
'X1000000.dat' u 1:2 w boxes lc rgb "#dddddd" t 'X1000000.dat' , 'X2000000.dat'
u 1:2 w boxes lc rgb "#77ff77" t 'X2000000.dat' , 'X2000001.dat' u 1:2 w boxes
lc "#ffaaaa" t 'X2000001.dat' , 'X2000002.dat' u 1:2 w boxes lc "#dddd55" t
'X2000002.dat' , 'X2000003.dat' u 1:2 w boxes lc rgb "#ffdddd" t 'X2000003.dat'
, 'X2000008.dat' u 1:2 w boxes lc rgb "#cc44ff" t 'X2000008.dat' ,
'X2000018.dat' u 1:2 w boxes lc rgb "#888888" t 'X2000018.dat' , 'X2000028.dat'
u 1:2 w boxes lc rgb "#bbddbb" t 'X2000028.dat' , 'X2000038.dat' u 1:2 w boxes
lc rgb "#ffee00" t 'X2000038.dat' , 'X2000048.dat' u 1:2 w boxes lc rgb
"#8844ff" t 'X2000048.dat' , 'X2000058.dat' u 1:2 w boxes lc rgb "#cceeff" t
'X2000058.dat' , 'X2000068.dat' u 1:2 w boxes lc rgb "#44bb44" t 'X2000068.dat'
, 'X2000078.dat' u 1:2 w boxes lc rgb "#99ee77" t 'X2000078.dat' ,
'X2000088.dat' u 1:2 w boxes lc rgb "#ffdd66" t 'X2000088.dat' , 'X2000098.dat'
u 1:2 w boxes lc rgb "#4444ff" t 'X2000098.dat' # histograma de p  (las dos
líneas siguientes alcanzan para graficar las p dentro del gnuplot) set key left
; set xrange[-3e-23:3e-23] p 'X0000500.dat' u 3:4 w boxes lc rgb "#dddddd" t
'X0000500.dat' , 'X0001000.dat' u 3:4 w boxes lc rgb "#77ff77" t 'X0001000.dat'
, 'X0002000.dat' u 3:4 w boxes lc "#ffaaaa" t 'X0002000.dat' , 'X0005000.dat' u
3:4 w boxes lc "#dddd55" t 'X0005000.dat' , 'X0010000.dat' u 3:4 w boxes lc rgb
"#ffdddd" t 'X0010000.dat' , 'X0020000.dat' u 3:4 w boxes lc rgb "#cc44ff" t
'X0020000.dat' , 'X0050000.dat' u 3:4 w boxes lc rgb "#888888" t 'X0050000.dat'
, 'X0100000.dat' u 3:4 w boxes lc rgb "#bbddbb" t 'X0100000.dat' ,
'X0200000.dat' u 3:4 w boxes lc rgb "#ffee00" t 'X0200000.dat' , 'X0500000.dat'
u 3:4 w boxes lc rgb "#8844ff" t 'X0500000.dat' , 'X0995000.dat' u 3:4 w boxes
lc rgb "#cceeff" t 'X0995000.dat' , 'X0999000.dat' u 3:4 w boxes lc rgb
"#44bb44" t 'X0999000.dat' , 'X0999500.dat' u 3:4 w boxes lc rgb "#99ee77" t
'X0999500.dat' , 'X1000000.dat' u 3:4 w boxes lc rgb "#ffdd66" t 'X1000000.dat'
, 'X2000000.dat' u 3:4 w boxes lc rgb "#4444ff" t 'X2000000.dat' set terminal qt


p 'X0000001.dat' u 1:2 w boxes lc rgb "#dddddd" t 'X0000001.dat' ,
'X0000100.dat' u 1:2 w boxes lc rgb "#77ff77" t 'X0000100.dat' , 'X0001000.dat'
u 1:2 w boxes lc "#ffaaaa" t 'X0001000.dat' , 'X0001200.dat' u 1:2 w boxes lc
"#dddd55" t 'X0001200.dat' , 'X0001400.dat' u 1:2 w boxes lc rgb "#ffdddd" t
'X0001400.dat' , 'X0001500.dat' u 1:2 w boxes lc rgb "#cc44ff" t 'X0001500.dat'
, 'X0001600.dat' u 1:2 w boxes lc rgb "#888888" t 'X0001600.dat' ,
'X0001700.dat' u 1:2 w boxes lc rgb "#bbddbb" t 'X0001700.dat' , 'X0001800.dat'
u 1:2 w boxes lc rgb "#ffee00" t 'X0001800.dat' , 'X0001900.dat' u 1:2 w boxes
lc rgb "#8844ff" t 'X0001900.dat' , 'X0002000.dat' u 1:2 w boxes lc rgb
"#cceeff" t 'X0002000.dat'

*/